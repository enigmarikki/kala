#include "hip/hip_runtime.h"
#include "rsw_solver.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <gmp.h>
#include "cgbn/cgbn.h"

#include <cstring>
#include <sstream>
#include <iomanip>

/* ---------- CUDA/CGBN error handling ---------- */
#define CUDA_CHECK(call)                                                      \
  do {                                                                        \
    hipError_t err = call;                                                   \
    if (err != hipSuccess) {                                                 \
      throw std::runtime_error(std::string("CUDA error: ") +                 \
                               hipGetErrorString(err));                      \
    }                                                                         \
  } while(0)

#define CGBN_CHECK(report)  cgbn_error_report_check(report)

/* ---------- CGBN parameters ---------- */
#define BITS 2048
#define TPI   32      /* threads-per-instance */

/* ---------- CGBN typedefs ---------- */
typedef cgbn_context_t<TPI>         context_t;
typedef cgbn_env_t<context_t,BITS>  env_t;
typedef env_t::cgbn_t               bn_t;

namespace rsw {

/* ---------- GPU instance structure ---------- */
struct gpu_inst {
    cgbn_mem_t<BITS> a, n, C;
    uint32_t         T;
    uint8_t          key[32];   /* output */
};

/* ---------- RSW kernel ---------- */
__global__ void rsw_kernel(cgbn_error_report_t *report,
                          gpu_inst *insts, int count) {
    
    int inst = (blockIdx.x * blockDim.x + threadIdx.x) / TPI;
    if (inst >= count) return;
    
    context_t ctx(cgbn_report_monitor, report, inst);
    env_t     env(ctx);
    
    bn_t a, n, C, res, k;
    cgbn_load(env, a, &insts[inst].a);
    cgbn_load(env, n, &insts[inst].n);
    cgbn_load(env, C, &insts[inst].C);
    
    /* Montgomery setup */
    uint32_t np0 = -cgbn_binary_inverse_ui32(env, cgbn_get_ui32(env, n));
    cgbn_bn2mont(env, res, a, n);
    
    /* 2^T sequential squarings */
    uint32_t T = insts[inst].T;
    for(uint32_t i = 0; i < T; i++)
        cgbn_mont_sqr(env, res, res, n, np0);
    
    cgbn_mont2bn(env, res, res, n, np0);  /* back to normal space */
    
    /* k = (C - res) mod n */
    if(cgbn_compare(env, C, res) >= 0)
        cgbn_sub(env, k, C, res);
    else {
        cgbn_add(env, k, C, n);
        cgbn_sub(env, k, k, res);
    }
    
    /* export 256-bit key little-endian */
    #pragma unroll
    for(int limb = 0; limb < 8; ++limb) {
        uint32_t w = cgbn_extract_bits_ui32(env, k, limb*32, 32);
        unsigned char *p = insts[inst].key + 4*limb;
        p[0] = w & 0xFF; 
        p[1] = (w >> 8) & 0xFF; 
        p[2] = (w >> 16) & 0xFF; 
        p[3] = (w >> 24) & 0xFF;
    }
}

/* ---------- Helper functions ---------- */
static void mpz_to_cgbn(cgbn_mem_t<BITS>& dst, const mpz_t src) {
    memset(&dst, 0, sizeof(dst));
    size_t cnt;
    mpz_export(dst._limbs, &cnt, -1, 4, 0, 0, src);
}

/* ---------- Implementation class ---------- */
class SolverImpl {
public:
    int device_id;
    hipDeviceProp_t device_props;
    cgbn_error_report_t *error_report;
    
    SolverImpl(int dev_id) : device_id(dev_id), error_report(nullptr) {
        CUDA_CHECK(hipSetDevice(device_id));
        CUDA_CHECK(hipGetDeviceProperties(&device_props, device_id));
        CUDA_CHECK(cgbn_error_report_alloc(&error_report));
    }
    
    ~SolverImpl() {
        if (error_report) {
            cgbn_error_report_free(error_report);
        }
    }
    
    size_t get_optimal_batch_size() const {
        // Heuristic based on GPU memory and compute capability
        size_t base_batch = 10000;
        if (device_props.major >= 7) {  // Volta and newer
            base_batch = 20000;
        }
        return base_batch;
    }
    
    SolveResult solve_single(const PuzzleParams& params) {
        SolveResult result;
        result.success = false;
        
        try {
            // Parse parameters
            mpz_t n, a, C;
            mpz_inits(n, a, C, nullptr);
            
            if (mpz_set_str(n, params.n, 16) != 0 ||
                mpz_set_str(a, params.a, 16) != 0 ||
                mpz_set_str(C, params.C, 16) != 0) {
                mpz_clears(n, a, C, nullptr);
                result.error_msg = "Invalid hex input";
                return result;
            }
            
            // Prepare GPU instance
            gpu_inst h_inst{};
            mpz_to_cgbn(h_inst.n, n);
            mpz_to_cgbn(h_inst.a, a);
            mpz_to_cgbn(h_inst.C, C);
            h_inst.T = params.T;
            
            // Allocate device memory
            gpu_inst *d_inst;
            CUDA_CHECK(hipMalloc(&d_inst, sizeof(gpu_inst)));
            CUDA_CHECK(hipMemcpy(d_inst, &h_inst, sizeof(gpu_inst), 
                                  hipMemcpyHostToDevice));
            
            // Launch kernel (1 instance)
            int threads = 128;
            int blocks = 1;
            rsw_kernel<<<blocks, threads>>>(error_report, d_inst, 1);
            
            CUDA_CHECK(hipDeviceSynchronize());
            CGBN_CHECK(error_report);
            
            // Get result
            CUDA_CHECK(hipMemcpy(&h_inst, d_inst, sizeof(gpu_inst), 
                                  hipMemcpyDeviceToHost));
            
            memcpy(result.key, h_inst.key, 32);
            result.success = true;
            
            // Cleanup
            CUDA_CHECK(hipFree(d_inst));
            mpz_clears(n, a, C, nullptr);
            
        } catch (const std::exception& e) {
            result.error_msg = e.what();
        }
        
        return result;
    }
    
    std::vector<SolveResult> solve_batch_impl(const std::vector<PuzzleParams>& params_batch) {
        std::vector<SolveResult> results(params_batch.size());
        
        if (params_batch.empty()) return results;
        
        try {
            // Prepare host batch
            std::vector<gpu_inst> h_batch(params_batch.size());
            
            for (size_t i = 0; i < params_batch.size(); i++) {
                mpz_t n, a, C;
                mpz_inits(n, a, C, nullptr);
                
                if (mpz_set_str(n, params_batch[i].n, 16) != 0 ||
                    mpz_set_str(a, params_batch[i].a, 16) != 0 ||
                    mpz_set_str(C, params_batch[i].C, 16) != 0) {
                    mpz_clears(n, a, C, nullptr);
                    results[i].success = false;
                    results[i].error_msg = "Invalid hex input";
                    continue;
                }
                
                mpz_to_cgbn(h_batch[i].n, n);
                mpz_to_cgbn(h_batch[i].a, a);
                mpz_to_cgbn(h_batch[i].C, C);
                h_batch[i].T = params_batch[i].T;
                
                mpz_clears(n, a, C, nullptr);
            }
            
            // Allocate device memory
            gpu_inst *d_batch;
            size_t batch_size = h_batch.size();
            CUDA_CHECK(hipMalloc(&d_batch, sizeof(gpu_inst) * batch_size));
            CUDA_CHECK(hipMemcpy(d_batch, h_batch.data(), 
                                  sizeof(gpu_inst) * batch_size, 
                                  hipMemcpyHostToDevice));
            
            // Calculate grid dimensions
            int threads = 128;
            int instances_per_block = threads / TPI;
            int blocks = (batch_size + instances_per_block - 1) / instances_per_block;
            
            // Launch kernel
            rsw_kernel<<<blocks, threads>>>(error_report, d_batch, batch_size);
            
            CUDA_CHECK(hipDeviceSynchronize());
            CGBN_CHECK(error_report);
            
            // Get results
            CUDA_CHECK(hipMemcpy(h_batch.data(), d_batch, 
                                  sizeof(gpu_inst) * batch_size, 
                                  hipMemcpyDeviceToHost));
            
            // Copy keys to results
            for (size_t i = 0; i < batch_size; i++) {
                if (results[i].error_msg.empty()) {
                    memcpy(results[i].key, h_batch[i].key, 32);
                    results[i].success = true;
                }
            }
            
            // Cleanup
            CUDA_CHECK(hipFree(d_batch));
            
        } catch (const std::exception& e) {
            for (auto& result : results) {
                if (!result.success && result.error_msg.empty()) {
                    result.error_msg = e.what();
                }
            }
        }
        
        return results;
    }
};

/* ---------- Solver implementation ---------- */
Solver::Solver(int device_id) : impl(std::make_unique<SolverImpl>(device_id)) {}

Solver::~Solver() = default;

Solver::Solver(Solver&&) noexcept = default;
Solver& Solver::operator=(Solver&&) noexcept = default;

SolveResult Solver::solve(const PuzzleParams& params) {
    return impl->solve_single(params);
}

std::vector<SolveResult> Solver::solve_batch(const std::vector<PuzzleParams>& params_batch) {
    return impl->solve_batch_impl(params_batch);
}

size_t Solver::get_optimal_batch_size() const {
    return impl->get_optimal_batch_size();
}

std::string Solver::get_device_name() const {
    return std::string(impl->device_props.name);
}

int Solver::get_device_id() const {
    return impl->device_id;
}

/* ---------- Utility functions ---------- */
namespace util {

std::vector<uint8_t> hex_to_bytes(const std::string& hex) {
    if (hex.size() % 2 != 0) {
        throw std::invalid_argument("Hex string must have even length");
    }
    
    std::vector<uint8_t> bytes(hex.size() / 2);
    for (size_t i = 0; i < bytes.size(); i++) {
        std::string byte_str = hex.substr(i * 2, 2);
        bytes[i] = static_cast<uint8_t>(std::stoi(byte_str, nullptr, 16));
    }
    return bytes;
}

std::string bytes_to_hex(const uint8_t* data, size_t len) {
    std::stringstream ss;
    ss << std::hex << std::setfill('0');
    for (size_t i = 0; i < len; i++) {
        ss << std::setw(2) << static_cast<int>(data[i]);
    }
    return ss.str();
}

} // namespace util
} // namespace rsw