#include "hip/hip_runtime.h"
/*********************************************************************
 *  RSW GPU timelock solver BENCHMARK - 10K puzzles
 *********************************************************************/
#include <hip/hip_runtime.h>
#include <gmp.h>
#include <hip/hip_runtime.h>
#include "cgbn/cgbn.h"

#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <chrono>
#include <vector>
#include <string>

#include <wolfssl/options.h>
#include <wolfssl/wolfcrypt/aes.h>

/* ---------- helpers: CUDA / CGBN error macros ---------- */
#define CUDA_CHECK(call)                                                      \
  do {                                                                        \
    hipError_t err = call;                                                   \
    if (err != hipSuccess) {                                                 \
      fprintf(stderr,"CUDA error %s:%d : %s\n",__FILE__,__LINE__,             \
              hipGetErrorString(err));                                       \
      exit(1);                                                                \
    }                                                                         \
  } while(0)

#define CGBN_CHECK(report)  cgbn_error_report_check(report)

/* ---------- parameters ---------- */
#define BITS 2048
#define TPI   32      /* threads‑per‑instance */
#define BATCH_SIZE 10000

/* ---------- CGBN typedefs ---------- */
typedef cgbn_context_t<TPI>         context_t;
typedef cgbn_env_t<context_t,BITS>  env_t;
typedef env_t::cgbn_t               bn_t;

/* ---------- GPU instance ---------- */
struct gpu_inst {
  cgbn_mem_t<BITS> a, n, C;
  uint32_t         T;
  uint8_t          key[32];   /* output */
};

/* ---------- kernel ---------- */
__global__ void rsw_kernel(cgbn_error_report_t *report,
                           gpu_inst *insts, int count) {

  int inst = (blockIdx.x * blockDim.x + threadIdx.x) / TPI;
  if (inst >= count) return;

  context_t ctx(cgbn_report_monitor, report, inst);
  env_t     env(ctx);

  bn_t a,n,C,res,k;
  cgbn_load(env,a,&insts[inst].a);
  cgbn_load(env,n,&insts[inst].n);
  cgbn_load(env,C,&insts[inst].C);

  /* Montgomery setup */
  uint32_t np0 = -cgbn_binary_inverse_ui32(env,cgbn_get_ui32(env,n));
  cgbn_bn2mont(env,res,a,n);

  /* 2^T sequential squarings */
  uint32_t T = insts[inst].T;
  for(uint32_t i=0;i<T;i++)
    cgbn_mont_sqr(env,res,res,n,np0);

  cgbn_mont2bn(env,res,res,n,np0);          /* back to normal space */

  /* k = (C - res) mod n */
  if(cgbn_compare(env,C,res)>=0)
    cgbn_sub(env,k,C,res);
  else {
    cgbn_add(env,k,C,n);
    cgbn_sub(env,k,k,res);
  }

  /* export 256‑bit key little‑endian */
  #pragma unroll
  for(int limb=0; limb<8; ++limb) {
    uint32_t w = cgbn_extract_bits_ui32(env,k, limb*32, 32);
    unsigned char *p = insts[inst].key + 4*limb;
    p[0]=w&0xFF; p[1]=(w>>8)&0xFF; p[2]=(w>>16)&0xFF; p[3]=(w>>24)&0xFF;
  }
}

/* ---------- helpers ---------- */
static std::vector<uint8_t> hex2vec(const char *s) {
  size_t L=strlen(s);  if(L%2){fprintf(stderr,"odd hex len\n");exit(1);}
  std::vector<uint8_t> v(L/2);
  for(size_t i=0;i<v.size();i++) sscanf(s+2*i,"%2hhx",&v[i]);
  return v;
}
static void hex2bytes(const char *s,uint8_t *out,size_t len){
  for(size_t i=0;i<len;i++) sscanf(s+2*i,"%2hhx",&out[i]);
}
static void mpz_set_hex(mpz_t z,const char* h){ if(mpz_set_str(z,h,16)) exit(1); }
static void mpz_to_cgbn(cgbn_mem_t<BITS>& dst,const mpz_t src){
  memset(&dst,0,sizeof(dst));
  size_t cnt; mpz_export(dst._limbs,&cnt,-1,4,0,0,src);
}

/* ---------- host AES-GCM decrypt ---------- */
static std::string aesgcm_decrypt_host(const uint8_t key[32],
                                       const uint8_t iv[12],
                                       const uint8_t tag[16],
                                       const std::vector<uint8_t>& ct)
{
    std::vector<uint8_t> pt(ct.size());
    Aes aes;
    
    int ret = wc_AesGcmSetKey(&aes, key, 32);
    if (ret != 0) {
        fprintf(stderr, "AesGcmSetKey failed: %d\n", ret);
        exit(1);
    }

    ret = wc_AesGcmDecrypt(&aes,
                           pt.data(),                 /* out */
                           ct.data(),  ct.size(),     /* in  */
                           iv, 12,
                           tag, 16,
                           nullptr, 0);               /* no AAD */

    if (ret != 0) {
        fprintf(stderr, "GCM auth fail %d\n", ret);
        exit(1);
    }
    return {reinterpret_cast<char*>(pt.data()), pt.size()};
}

/* ---------- main ---------- */
int main(int argc,char**argv){
  if(argc!=8){
    fprintf(stderr,"usage: %s n a C T iv ct tag (hex)\n",argv[0]); return 1;}

  mpz_t n,a,C; mpz_inits(n,a,C,nullptr);
  mpz_set_hex(n,argv[1]); mpz_set_hex(a,argv[2]); mpz_set_hex(C,argv[3]);
  uint32_t T=strtoul(argv[4],nullptr,10);

  /* parse crypto params */
  uint8_t iv[12], tag[16];
  hex2bytes(argv[5], iv, 12);
  hex2bytes(argv[7], tag, 16);
  std::vector<uint8_t> ct = hex2vec(argv[6]);

  /* prepare single puzzle instance */
  gpu_inst single{}; 
  mpz_to_cgbn(single.n,n); 
  mpz_to_cgbn(single.a,a); 
  mpz_to_cgbn(single.C,C); 
  single.T=T;

  /* allocate for 10K instances (all same puzzle) */
  gpu_inst *h_batch = new gpu_inst[BATCH_SIZE];
  for(int i = 0; i < BATCH_SIZE; i++) {
    h_batch[i] = single;  // copy same puzzle
  }

  gpu_inst *d_batch; 
  cgbn_error_report_t *report;
  
  CUDA_CHECK(hipSetDevice(0));
  CUDA_CHECK(hipMalloc(&d_batch, sizeof(gpu_inst) * BATCH_SIZE));
  CUDA_CHECK(hipMemcpy(d_batch, h_batch, sizeof(gpu_inst) * BATCH_SIZE, hipMemcpyHostToDevice));
  CUDA_CHECK(cgbn_error_report_alloc(&report));

  /* calculate grid dimensions */
  int threads = 128;
  int instances_per_block = threads / TPI;
  int blocks = (BATCH_SIZE + instances_per_block - 1) / instances_per_block;

  printf("Solving %d puzzles...\n", BATCH_SIZE);
  printf("Grid: %d blocks x %d threads\n", blocks, threads);
  printf("Instances per block: %d\n", instances_per_block);

  /* warm up */
  rsw_kernel<<<blocks,threads>>>(report, d_batch, BATCH_SIZE);
  CUDA_CHECK(hipDeviceSynchronize());

  /* actual benchmark */
  auto t0 = std::chrono::high_resolution_clock::now();
  
  rsw_kernel<<<blocks,threads>>>(report, d_batch, BATCH_SIZE);
  CUDA_CHECK(hipDeviceSynchronize());
  
  auto t1 = std::chrono::high_resolution_clock::now();
  
  CGBN_CHECK(report);

  /* get results back */
  CUDA_CHECK(hipMemcpy(h_batch, d_batch, sizeof(gpu_inst) * BATCH_SIZE, hipMemcpyDeviceToHost));

  /* timing stats */
  auto total_ms = std::chrono::duration_cast<std::chrono::milliseconds>(t1-t0).count();
  double ms_per_puzzle = (double)total_ms / BATCH_SIZE;
  double puzzles_per_sec = 1000.0 / ms_per_puzzle;

  printf("\n===== BENCHMARK RESULTS =====\n");
  printf("Total time: %ld ms\n", total_ms);
  printf("Time per puzzle: %.3f ms\n", ms_per_puzzle);
  printf("Throughput: %.1f puzzles/sec\n", puzzles_per_sec);
  printf("=============================\n");

  /* verify first and last results match */
  printf("\nFirst key:  ");
  for(int i=0; i<32; i++) printf("%02x", h_batch[0].key[i]);
  printf("\nLast key:   ");
  for(int i=0; i<32; i++) printf("%02x", h_batch[BATCH_SIZE-1].key[i]);
  printf("\n");

  /* decrypt the message */
  std::string plaintext = aesgcm_decrypt_host(h_batch[0].key, iv, tag, ct);
  printf("\nDecrypted message: \"%s\"\n", plaintext.c_str());

  /* cleanup */
  CUDA_CHECK(hipFree(d_batch)); 
  CUDA_CHECK(cgbn_error_report_free(report));
  delete[] h_batch;
  mpz_clears(n,a,C,nullptr);
  
  return 0;
}